// Compile with nvcc 2dconvec.cu -lcublas -arch sm_13
// -lcublas because we're using cublas
// -arch sm_13 to alert the compiler that we're using a GPU that supports
// float-precision.
// tab spacing = 4

// NOTICE
// This file was stored by David Sanchez in 2023, based on a recovered hard-drive.  The code almost certainly dates to 2010 (earlier/later).
// I think I wrote this based on Matlab code by Grady Wright and Greg Barnett, under the supervision of Dave Yuen (and advice + comments from many).
// That said, I do not recall exactly when or under what conditions this was written and it may have been given to me.
// In whole, in part, or in derivative this code forms the basis of some papers, but I've lost track of which ones.  I don't even know whether this
// is the most up-to-date such code.
//
// Sincere apologies.  Good luck.



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <time.h>
#include <string.h>
#include "hipblas.h"

// Parameters.  These may change from run-to-run
#define M (500) // vertical dimension of the temperature array
#define RA (100000000.0) // Rayleigh number
#define XF (2)	// Aspect ratio for the temperature array
#define DBG_ON 0 // Used to toggle writing to the debug file.  Slow!
#define PRINTVORT (1) // Write the vorticity to file
#define PRINTVEL (1) // Write to the convecvelocity.dat file.
#define PRINTSTREAM (1) // Write the streamfunction to file.
#define PRINTT (1) // Write the temperature data to file
#define PRINTNU (1) // Write the Nusselt number data to file.
#define STARTSTEP (0) // First step.  Useful if filenames are to be consistent
					  // and INPUTT is on
#define ENDSTEP (1000000) // Last timestep to compute.  If
						 // ENDSTEP%FRAMSIZE != 0 don't expect this to be nice
#define INPUTT (0) // Read the value of T from input2d.bin
#define INPUTT32 (0) // Read the value of T from input2d32.bin, filled with floats instead of uint8_t
#define BSNAME "ra108_1000x498_jul27_" // for filenames.  BE CAREFUL WITH IT

// Constants.  Stability cannot be assured if these values are altered.
//#define N (XF*(M-1)+1) // horizontal dimension of temperature array
#define N (1000)
#define DX (1/(M - 1.0)) // x and z-dimensional mesh spacing
#define DX2 (DX*DX) // DX^2, for saving a few computations.
#define OMEGACOEFF (-((DX*DX)*(DX*DX))*RA) // Used on every timestep

#define PI 3.1415926535897932384626433832795028841968 // Precision here is arbitrary and may be traded
							 // for performance if context allows
#define DT_START 0.000000000000005 //timestep width.  Needs to be small to
								    //guide the model through high Ra runs.


#define FRAMESIZE (DX2/4.0) // How many iterations between saves
// flatten a 2D grid with 1D blocks into a vector.  The functionality
// could be extended to perform pointer arithmetic, but that's not necessary
// in this code.
//
//  Invoked grid geometry
//  (2D grid with 1D blocks)
//
//  |---|---|---|
//  |1,1|1,2|1,3|
//  |---|---|---|
//  |2,1|2,2|2,3|  ==>  1,1 1,2 1,3 2,1 2,2 ...
//  |---|---|---|
//  |3,1|3,2|3,3|
//  |---|---|---|


#define THREADID (((gridDim.x)*(blockIdx.y) + (blockIdx.x))*(blockDim.x) + (threadIdx.x))

// It is possible to alternate between 2D row-major and column-major formats by
// taking transposes.
#define TPOSE(i,j,ld) ((j)*(ld) + (i))

// Simplify device-side array allocation, assumes all types are float.
#define CUALLOC(elements, name) float* name; custat = cublasAlloc(elements, sizeof(float), (void**)&name); if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n")

// Simplify calling G, since many of the arguments are assured
#define SHORTG(input, compute, save, frames) G(input, d_Tbuff, d_DxT, d_y, d_u, d_v, d_psi, d_omega, d_dsc, d_dsr, d_ei, d_dt, save, h_T, compute, frames, tstep)

// Define a global N-by-N array h_DBG, for debugging purposes.
float h_DBG[N*N];

//-----------------------------------------------------------------------------
//=============================================================================
//									KERNELS
//=============================================================================
//-----------------------------------------------------------------------------

//=============================================================================
//							   ElementMultOmega
//=============================================================================
// Performs elementwise matrix multiplication on matrices shaped like omega,
// returning the result in A.
__global__ void ElemMultOmega(float* A, float* B) {
	if(THREADID < (M-2)*(N-2) ) {
		A[THREADID] = A[THREADID]*B[THREADID];
	}
	return;
}

//=============================================================================
//								 ElementMultT
//=============================================================================
// Performs elementwise matrix multiplication on matrices shaped like T,
// putting the result in A.
__global__ void ElemMultT(float* A, float* B) {
	if(THREADID < (M-2)*(N)) {
		A[THREADID] = A[THREADID]*B[THREADID];
	}
	return;
}

//=============================================================================
//								 ElementMultNu
//=============================================================================
// Performs elementwise matrix multiplication on matrices shaped like d_nutop,
// putting the result in A.
__global__ void ElemMultNu(float* A, float* B) {
	if(THREADID < N) {
		A[THREADID] = A[THREADID]*B[THREADID];
	}
	return;
}

//=============================================================================
//								    SubOne
//=============================================================================
// Subtracts 1.0 from every element in a vector (floats) of length N
__global__ void SubOne(float* A) {
	if(THREADID < N) {
		A[THREADID] = A[THREADID] - 1.0;
	}
	return;
}

//=============================================================================
//								    AddOne
//=============================================================================
// Adds 1.0 from every element in a vector (floats) of length N
__global__ void AddOne(float* A) {
	if(THREADID < N) {
		A[THREADID] = A[THREADID] + 1.0;
	}
	return;
}

//=============================================================================
//								    AddX
//=============================================================================
// Adds x from every element in a vector (floats) of length (M-2)*N
__global__ void AddX(float* A, float x) {
	if(THREADID < (M-2)*N) {
		A[THREADID] = A[THREADID] + x;
	}
	return;
}

//=============================================================================
//								   Updatedt
//=============================================================================
// Adaptive update rule for dt.  d_dt (a device-side one-element array) should
// be passed as dt, whereas ptru and ptrv point (1-indexed) to vectors u and v.
// The current value of dt[0] will be overwritten. Can be called with a 1D
// grid containing a single 1D block with one thread.
__global__ void Updatedt(int ptru, float* u, int ptrv, float* v, float* dt) {
	if( (threadIdx.x + blockIdx.x*blockDim.x) == 0) {
		dt[0] = max(abs(u[ptru - 1]),abs(v[ptrv - 1]));
		dt[0] = min(DX/dt[0],DX2/4.0);
	}
	return;
}
//-----------------------------------------------------------------------------
//=============================================================================
//							  Host-Side Routines
//=============================================================================
//-----------------------------------------------------------------------------

//=============================================================================
//								   WriteT
//=============================================================================
// WriteT will write 255 times the uint8 equivalent of the float x.  Why?
// Because this is the perfect format for .bob files.
void WriteT(float x, FILE* outfile) {
	uint8_t savnum = 255*x;
	// C typecasts down by truncation, so adding 0.5 forces it to round.
	// Inverse for negatives.
	if(x >= 0) {
		savnum = (uint8_t)(255*x + 0.5);
	}
	else {
		savnum = (uint8_t)(255*x - 0.5);
	}
	fwrite(&savnum, 1, 1, outfile);
	return;
}
//=============================================================================
//                                 WriteT32
//=============================================================================
// WriteT will write T in IEEE 32-bit form
void WriteT32(float x, FILE* outfile) {
        fwrite(&x, 4, 1, outfile);
        return;
}


//=============================================================================
//								   PrintGPU
//=============================================================================
// Takes a matrix (of floats) on the GPU and prints it to file.
//void PrintGPU(float* f, int rows, int cols) {
//	cublasGetVector(rows*cols, sizeof(float), &f[0], 1, &h_DBG[0], 1);
//	for(int i = 0; i < rows; i++) {
//		for(int j = 0; j < cols; j++) {
//			fprintf(dbgfile, "%2.4f ", h_DBG[i*cols + j]);
//		}
//		fprintf(dbgfile, "\n");
//	}
//	return;
//}

//=============================================================================
//									NormalizeT
//=============================================================================
// Normalizes the values of T to [0,1] as a safeguard against explosion
void NormalizeT(float* T) {
	float maxval[1];
	float minval[1];
	hipblasGetVector(1, sizeof(float), T + (cublasIsamax(N*(M-2), T, 1) - 1), 1, maxval, 1);
	hipblasGetVector(1, sizeof(float), T + (cublasIsamin(N*(M-2), T, 1) - 1), 1, minval, 1);
	AddX<<<floor((N*(M-2))/256.0) + 1, 256>>>(T, -minval[0]);
	cublasSscal(N*(M-2), 1/(maxval[0] - minval[0]), T, 1);
	return;
}


//=============================================================================
//									NusseltCompute
//=============================================================================
// Returns the Nusselt number of the array T, which is pointed to in GPU space
float NusseltCompute(float* T, float* nutop, float* ztop, float* zbot, float* nubot, float* trnu) {
		float topsum, botsum;
		// Calculate the Nusselt number along the top of the array.
		// d_nutop is the last three rows of T, in inverse order,
		// with all 0s along the bottom.

		// Copy the last three rows of T into the first three rows of nutop
		cublasScopy(N, (T + (M-5)*N), 1, (nutop), 1);
		cublasScopy(N, (T + (M-4)*N), 1, (nutop + (N)), 1);
		cublasScopy(N, (T + (M-3)*N), 1, (nutop + (2*N)), 1);

		// Set the last row of nutop = 0.
		cublasSscal(N, 0.0, (nutop + (3*N)), 1);

		// nutop += -( 1 - ztop)
		// => nutop += ztop; nutop -= 1
		cublasSaxpy(4*N, 1.0, ztop, 1, nutop, 1);
		// Subtract 1 from every element in the array.  SubOne works on rows.
		SubOne<<<floor(N/256.0) + 1, 256>>>(nutop);
		SubOne<<<floor(N/256.0) + 1, 256>>>(nutop + N);
		SubOne<<<floor(N/256.0) + 1, 256>>>(nutop + 2*N);
		SubOne<<<floor(N/256.0) + 1, 256>>>(nutop + 3*N);

		// -(2/3)*row0 + 3*row1 - 6*row2 + (11/3)*row3
		// accumulate in the 0th row
		// scale the 0th row by -(2/3)
		cublasSscal(N, -(2.0/3.0), nutop, 1);
		// Add 3*row1
		cublasSaxpy(N, 3.0, (nutop + N), 1, nutop, 1);
		// Add - 6*row2
		cublasSaxpy(N, -6.0, (nutop + (2*N)), 1, nutop, 1);
		// Add (11/3)*row3
		cublasSaxpy(N, (11.0/3.0), (nutop + (3*N)), 1, nutop, 1);
		// Divide the array by 2*DX
		cublasSscal(4*N, 1/(2.0*DX), nutop, 1);
		// Elementwise multiplication with trnu
		ElemMultNu<<<floor(N/256.0) + 1, 256>>>(nutop, trnu);
		// Sum up the elements of row0, by performing a dot product with
		// a row that has been altered to be all 1s.
		// Empty row1, then add 1 to all its elements
		cublasSscal(N, 0.0, (nutop + N), 1);
		AddOne<<<floor(N/256.0) + 1, 256>>>(nutop + N);

		topsum = cublasSdot(N, nutop, 1, (nutop + N), 1)/(-XF);

		// Calculate the Nusselt number along the bottom of the array.
		// d_nubot's first row is all 1, and ith row is the i-1th row of d_T
		// Put the first row of T in nubot, then subtract to get 0, then AddOne
		cublasSscal(N, 0.0, nubot, 1);
		AddOne<<<floor(N/256.0) + 1, 256>>>(nubot);
		cublasScopy(N, T, 1, (nubot + N), 1);
		cublasScopy(N, (T + N), 1, (nubot + 2*N), 1);
		cublasScopy(N, (T + 2*N), 1, (nubot + 3*N), 1);

		// nubot += -( 1 - zbot)
		// => nubot += zbot; nubot -= 1
		cublasSaxpy(4*N, 1.0, zbot, 1, nubot, 1);
		// Subtract 1 from every element in the array.  SubOne works on rows.
		SubOne<<<floor(N/256.0) + 1, 256>>>(nubot);
		SubOne<<<floor(N/256.0) + 1, 256>>>(nubot + N);
		SubOne<<<floor(N/256.0) + 1, 256>>>(nubot + 2*N);
		SubOne<<<floor(N/256.0) + 1, 256>>>(nubot + 3*N);

		// -(11/3)*row0 + 6*row1 - 3*row2 + (2/3)*row3
		// accumulate in the 0th row
		// scale the 0th row by -(11/3)
		cublasSscal(N, -(11.0/3.0), nubot, 1);
		// Add 6*row1
		cublasSaxpy(N, 6.0, (nubot + N), 1, nubot, 1);
		// Add -3*row2
		cublasSaxpy(N, -3.0, (nubot + (2*N)), 1, nubot, 1);
		// Add (2/3)*row3
		cublasSaxpy(N, (2.0/3.0), (nubot + (3*N)), 1, nubot, 1);
		// Divide the array by 2*DX
		cublasSscal(4*N, 1/(2.0*DX), nubot, 1);
		// Elementwise multiplication with trnu
		ElemMultNu<<<floor(N/256.0) + 1, 256>>>(nubot, trnu);
		// Sum up the elements of row0, by performing a dot product with
		// a row that has been altered to be all 1s.
		// The second row of nutop has already been set up for this.
		botsum = cublasSdot(N, nubot, 1, (nutop + N), 1)/(-XF);

		return(topsum);
}
//=============================================================================
//									  Dz
//=============================================================================
// Finite-difference approximation to the first derivative with respect to z of
// a matrix shaped like T.  Execution forks if f is known to be psi.  Uses only
// row manipulations and the subtraction of 1 from each element of a vector. To
// extract a row, cublas routines are used.  The elements of the first column
// are separated in memory by N elements, so the initial elements of each row
// are likewise separated.  The individual elements of a single row are
// separated in memory by 1 element.

void Dz(float* f, int is_it_psi, float* y) {
	// yrows[i] = frows[i + 1] - frows[i - 1]
	// Move all but the first two rows of f into the interior rows of y.
	// The end of one row is one element away from the beginning of the next,
	// so adjacent rows are laid out in memory identically to a vector.
	cublasScopy(N*(M-4), (f + (2*N)), 1, (y + N), 1);
	// Subtract all but the last two rows of f.
	cublasSaxpy(N*(M-4), -1.0, f, 1, (y + N), 1);

	if(is_it_psi == 1) {
		// yrows[0] = frows[1]
		// Move the second row of f into the first row of y
		cublasScopy(N, (f + N), 1, y, 1);
	}
	else {
		// yrows[0] = frows[1] - 1
		// Move the second row of f into the first row of y
		cublasScopy(N, (f + N), 1, y, 1);
		// Subtract 1 from every element in the first row of y.
		SubOne<<<floor(N/256.0) + 1, 256>>>(y);

	}

	// yrows[M-3] = -frows[M-4]
	// Copy the second-to-last row of f into the last row of y
	cublasScopy(N, (f + ((M - 4)*N)), 1, (y + (M - 3)*N), 1);
	// Scale by -1.0
	cublasSscal(N, -1.0, (y + (M - 3)*N), 1);
	// Scale y by 1/(2*DX)
	cublasSscal(N*(M-2), 1.0/(2.0*DX), y, 1);
	return;
}

//=============================================================================
//									  Dzz
//=============================================================================
// Finite-difference approximation to the second derivative with respect to z
// of a  T-shaped array.  Uses only row manipulations and the addition of 1 to
// each element of a vector.  To extract a row, cublas routines are used.  The
// elements of the first column are separated in memory by N elements, so the
// initial elements of each row are likewise separated.  The individual
// elements of a single row are separated in memory by 1 element.

void Dzz(float* f, float* y) {
	// yrows[i] = frows[i - 1] - 2*frows[i] + frows[i + 1]
	// Move all but the last two rows of f into the interior rows of y.
	cublasScopy(N*(M-4), f, 1, (y + N), 1);

	// Subtract 2* the interior rows of f
	cublasSaxpy(N*(M-4), -2.0, (f + N), 1, (y + N), 1);

	// Add all but the first two rows of f.
	cublasSaxpy(N*(M-4), 1.0, (f + (2*N)), 1, (y + N), 1);

	// yrows[0] = 1 - 2*frows[0] + frows[1]
	// Copy the first row of f into the first row of y
	cublasScopy(N, f, 1, y, 1);

	// scale by -2
	cublasSscal(N, -2.0, y, 1);

	// add the second row of f
	cublasSaxpy(N, 1.0, (f + N), 1, y, 1);

	// Add 1 to every element in the first row of y.
	AddOne<<<floor(N/256.0) + 1, 256>>>(y);

	// yrows[M-3] = frows[M-4] - 2*frows[M-3]
	// move the second-to-last row of f into the last row of y
	cublasScopy(N, (f + (M - 4)*N), 1, (y + (M - 3)*N), 1);
	// subtract 2* the last row of f
	cublasSaxpy(N, -2.0, (f + (M - 3)*N), 1, (y + (M - 3)*N), 1);
	// Scale y by 1/DX2
	cublasSscal((M-2)*N, (1.0/DX2), y, 1);
	return;
}

//=============================================================================
//									  Dx
//=============================================================================
// Finite difference approximation to the first derivative with
// respect to x of a T-shaped matrix.  Forks if f is known to be psi. Uses only
// column manipulations and assumes all matrices are in row-major.  To extract
// a column, cublas routines are used.  If the beginning of an array is at f,
// then the elements of the first row (start of each column) are separated by
// one element, and each element within a column is separated by the length
// of a row, N.

void Dx(float* f, int is_it_psi, float* y) {
	// ycols[i] = fcols[i+1] - fcols[i-1], interior cols
	// Copy all but the first two columns of f into the interior columns of y.
	// Copy row-by-row instead of column-by-column, since Dcopy is optimized
	// for longer vectors.

	for(int i = 0; i < M - 2; i++) {
		cublasScopy((N-2), (f + i*N) + 2, 1, (y + i*N) + 1, 1);
	}
	// Subtract the block corresponding to all but the last two columns of f.
	for(int i = 0; i < M - 2; i++) {
		cublasSaxpy((N-2), -1.0, (f + i*N), 1, (y + i*N) + 1, 1);
	}

	if(is_it_psi == 1) {
		// ycols[0] = 6*fcols[1] - 3*fcols[2] + (2/3)*fcols[3]
		// Begin by copying the second column of f into the first column of y
		cublasScopy((M-2), (f + 1), N, y, N);
		// Scale it by a factor of 6
		cublasSscal((M-2), 6.0, y, N);
		// Subtract the third column of 3*f
		cublasSaxpy((M-2), -3.0, (f + 2), N, y, N);
		// Add the fourth column of (2/3)*f
		cublasSaxpy((M-2), (2.0/3.0), (f + 3), N, y, N);

		//ycols[N-1] = -6*fcols[N-2] + 3*fcols[N-3] - (2/3)*fcols[N-4]
		// Copy the second-to-last column of f into the last column of y
		cublasScopy((M-2), (f + (N - 2)), N, (y + (N - 1)), N);
		// Scale it by a factor of -6
		cublasSscal((M-2), -6.0, (y + (N - 1)), N);
		// Add the third-to-last column of 3*fcols[N-3]
		cublasSaxpy((M-2), 3.0, (f + (N - 3)), N, (y + (N - 1)), N);
		// Subtract the fourth-to-last column of (2/3)*f[N-4]
		cublasSaxpy((M-2), -(2.0/3.0), (f + (N - 4)), N, (y + (N - 1)), N);
	}
	else {
		// outside columns = 0
		cublasSscal((M-2), 0.0, y, N);
		cublasSscal((M-2), 0.0, (y + (N - 1)), N);

	}

	// Scale y by (1/(2*DX))
	cublasSscal(N*(M-2), (1.0/(2.0*DX)), y, 1);
	return;
}

//=============================================================================
//									  Dxx
//=============================================================================
// Finite-difference approximation to the second derivative with
// respect to x of a T-shaped matrix.  The input is always going to be a temp-
// erature array.  Uses only column manipulations and assumes all matrices are
// in row-major.  To extract a column, cublas routines are used.  If the
// beginning of an array is at f, then the elements of the first row (start of
// each column) are separated by one element, and each element within a column
// is separated by the length of a row, N.

void Dxx(float* f, float* y) {
	// ycols[i] = fcols[i-1] - 2*fcols[i] + fcols[i+1], interior columns
	// copy f into y
	cublasScopy(N*(M-2), f, 1, y, 1);
	// scale by -2
	cublasSscal(N*(M-2), -2.0, y, 1);
	// Add the block corresponding to all but the last two columns of f.
	for(int i = 0; i < M-2; i++) {
		cublasSaxpy((N-2), 1.0, (f + i*N), 1, (y + i*N) + 1, 1);
	}
	// Add the block corresponding to all but the first two columns of f.
	for(int i = 0; i < M-2; i++) {
		cublasSaxpy((N-2), 1.0, (f + i*N) + 2, 1, (y + i*N) + 1, 1);
	}

	// ycols[0] = -2*fcols[0] + 2*fcols[1]
	// Copy the first column of f into the first column of y.
	cublasScopy((M-2), f, N, y, N);
	// Scale the first column of y by -2.0
	cublasSscal((M-2), -2.0, y, N);
	// Add 2* the second column of f.
	cublasSaxpy((M-2), 2.0, (f + 1), N, y, N);

	// ycols[N-1] = -2*fcols[N-1] + 2*fcols[N-2]
	// Move the last column of f into the last column of y
	cublasScopy((M-2), (f + (N - 1)), N, (y + (N - 1)), N);
	// Scale by -2.0
	cublasSscal((M-2), -2.0, (y + (N - 1)), N);
	// add 2* the second-to-last column of f.
	cublasSaxpy((M-2), 2.0, (f + (N - 2)), N, (y + (N - 1)), N);

	// Scale y by 1/(DX^2)
	cublasSscal(N*(M-2), 1.0/(DX2), y, 1);
	return;
}



//=============================================================================
//									   G
//=============================================================================
// Computes the RK1 approximation using finite difference method, storing the
// result in output
void G(float* f,
	   float* Tbuff,
	   float* DxT,
	   float* y,
	   float* u,
	   float* v,
	   float* psi,
	   float* omega,
	   float* dsc,
	   float* dsr,
	   float* ei,
	   float* dt,
	   float* output,
	   float* h_T,
	   int compute_velocity,
	   int frames,
	   int tstep) {

	// Define the grid dimensions
	dim3 grid(floor(N/16) + 1, floor((M-2)/16) + 1), block(256);

	// Define omega to be the interior columns of Dxf
	// Save Dx of f in DxT for later
	cublasScopy(N*(M-2), f, 1, DxT, 1);
	Dx(f, 0, DxT);

	// Copy the interior columns of DxT to omega
	for(int i = 0; i < M-2; i ++) {
		cublasScopy((N-2), (DxT + i*N)+1, 1, (omega + i*(N-2)), 1);
	}

	// Perform some matrix multiplications.  cublas assumes everything is in
	// column-major, so while we want to perform:
	// omega = dsc*omega
	// omega = omega*dsr
	// omega = omega.*ei
	// omega = dsc*omega
	// omega = omega*dsr
	// we observe that Transpose(A*B) = Transpose(B)*Transpose(A) to perform
	// these same manipulations while preserving row-major storage.
	// Omega has dimensions (M-2)xN, but cublas thinks this is Nx(M-2).
	// dsc and dsr are square..
	// Perform Transpose(omega) = Transpose(omega)*Transpose(dsc):
	//
	//					 _M-2_
	//					|	  |
	//				M-2	| dsc |
	//					|_____| ld = M-2
	//
	//		 _M-2_		 _M-2_
	//		|	  |		|	  |
	//		|	  |		|	  |
	//		|	  |		| new |
	//	 N-2|omega|	 N-2|omega|
	//		|	  |		|	  |
	//		|ld=N-2		|ld=N-2
	//		|_____|		|_____|
	// The call for cublasSgemm is ('n', 'n', m, n, k, alpha, A, lda, B, ldb,
	// beta, C, ldc), where A is m-by-k, B is k-by-n, and C is m-by-n.
	// Since A = C, k = n, so B is k-by-k.  B = dsc, so k=n=M-2 and m = N-2.
	cublasSgemm('n', 'n', N-2, M-2, M-2, 1.0, omega, N-2, dsc, M-2, 0.0, Tbuff, N-2 );
	cublasScopy((N-2)*(M-2), Tbuff, 1, omega, 1);

	// Perform Tranpose(dsr)*Transpose(omega), store in omega.
	//						 _M-2_
	//						|	  |
	//						|	  |
	//						| 	  |
	//					 N-2|omega|
	//						|	  |
	//						|ld=N-2
	//						|_____|
	//
	//		 ___N-2___		 _M-2_
	//		|		  |		|	  |
	//		|		  |		|	  |
	//		|		  |		| new |
	//	 N-2|	dsr   |	 N-2|omega|
	//		|		  |		|	  |
	//		|  ld=N-2 |		|ld=N-2
	//		|_________|		|_____|
	// since A is square, M = k = N-2, so n must be M-2.
	cublasSgemm('n', 'n', N-2, M-2, N-2, 1.0, dsr, N-2, omega, N-2, 0.0, Tbuff, N-2);
	cublasScopy((N-2)*(M-2), Tbuff, 1, omega, 1);

	// elementwise matrix multiplication, storing the result in omega
//DEBUG
	ElemMultOmega<<<grid, block>>>(omega, ei);

	// same Transpose(omega)*Transpose(dsc) operation as before
	cublasSgemm('n', 'n', N-2, M-2, M-2, 1.0, omega, N-2, dsc, M-2, 0.0, Tbuff, N-2);
	cublasScopy((N-2)*(M-2), Tbuff, 1, omega, 1);


	// same Transpose(dsr)*Transpose(omega) operation as before
	cublasSgemm('n', 'n', N-2, M-2, N-2, 1.0, dsr, N-2, omega, N-2, 0.0, Tbuff, N-2);
	cublasScopy((N-2)*(M-2), Tbuff, 1, omega, 1);

	if(PRINTVORT == 1 && frames > FRAMESIZE ) {
		char stringnum[25];
		float x;
		char vortname[100] = "Vort" BSNAME;
		sprintf(stringnum,"%d.bin",tstep);
		strcat(vortname,stringnum);
		// Open a stream for writing Vort data
		FILE *vortfile = fopen(vortname,"w");
		hipblasGetVector((N-2)*(M-2), sizeof(float), omega, 1, h_T, 1);
		// We have to normalize vorticity, but its sign is very important.
		float maxx = h_T[0];
		float minx = h_T[0];
		for(int k = 0; k < (N-2)*(M-2); k++) {
			if(minx > h_T[k]) minx = h_T[k];
			if(maxx < h_T[k]) maxx = h_T[k];
		}
		for(int i = M-3; i > -1; i--) {
        	for(int j = 0; j < N-2; j++) {
				x = (float)h_T[i*(N-2)+j];
//				if(x < 0) x /= minx;
//				if(x > 0) x /= maxx;
//				x += 1.0;
//				x /= 2.0;
				WriteT32(h_T[i*(N-2)+j], vortfile);
			}
		}
		fclose(vortfile);
	}

	// Scale omega by -(DX^4)*(RA) = OMEGACOEFF
	cublasSscal((N-2)*(M-2), OMEGACOEFF, omega, 1);

	// interior columns of psi = (RA*DX^4)*omega
	// copy omega into the interior columns of psi
	// omega has rows of length N-2 instead of N

	for(int i = 0; i < M-2; i ++) {
		cublasScopy((N-2), (omega + i*(N-2)), 1, (psi + i*N)+1, 1);
	}

	// Write streamfunction to file
	if(PRINTSTREAM == 1 && frames > FRAMESIZE) {
		char stringnum[25];
		float x;
		char streamname[100] = "Stream" BSNAME;
		sprintf(stringnum,"%d.bin",tstep);
		strcat(streamname,stringnum);
		// Open a stream for writing streamfunction data
		FILE *streamfile = fopen(streamname,"w");
		hipblasGetVector(N*(M-2), sizeof(float), psi, 1, h_T, 1);
		// We have to normalize vorticity, but its sign is very important.
		float maxx = h_T[0];
		float minx = h_T[0];
		for(int k = 0; k < N*(M-2); k++) {
			if(minx > h_T[k]) minx = h_T[k];
			if(maxx < h_T[k]) maxx = h_T[k];
		}
		for(int i = M-3; i > -1; i--) {
                        for(int j = 0; j < N; j++) {
                                x = (float)h_T[i*N+j];
                                if(x < 0) x /= minx;
                                if(x > 0) x /= maxx;
                                x += 1.0;
                                x /= 2.0;
                                WriteT32(x, streamfile);
                        }
                }
		fclose(streamfile);
	}
	// Velocity in the x-direction
	cublasScopy(N*(M-2), f, 1, u, 1);
	Dz(psi, 1, u);

	if(PRINTVEL == 1 && frames > FRAMESIZE) {
		char stringnum[25];
		float x;
		char vxname[100] = "Vx" BSNAME;
		sprintf(stringnum,"%d.bin",tstep);
		strcat(vxname,stringnum);
		// Open a stream for writing Vx data
		FILE *vxfile = fopen(vxname,"w");
		hipblasGetVector(N*(M-2), sizeof(float), u, 1, h_T, 1);
                for(int i = M-3; i > -1; i--) {
                	for(int j = 0; j < N; j++) {
                        x = (float)h_T[i*N+j];
						fwrite(&x,4,1,vxfile);
                	}
                }
		fclose(vxfile);
        }

	// v is -Dxpsi, velocity in the z direction.
	// Place Dxpsi into v
	cublasScopy(N*(M-2), f, 1, v, 1);
	Dx(psi, 1, v);

        if(PRINTVEL == 1 && frames > FRAMESIZE) {
                char stringnum[25];
		float x;
                char vyname[100] = "Vy" BSNAME;
                sprintf(stringnum,"%d.bin",tstep);
                strcat(vyname,stringnum);
                // Open a stream for writing Vy data
                FILE *vyfile = fopen(vyname,"w");
                hipblasGetVector(N*(M-2), sizeof(float), v, 1, h_T, 1);
                for(int i = M-3; i > -1; i--) {
                        for(int j = 0; j < N; j++) {
                        	x = (float)h_T[i*N+j];
                            fwrite(&x,4,1,vyfile);
                        }
                }
		fclose(vyfile);
        }

	// Change the sign of v.
	cublasSscal((M-2)*N, -1.0f, v, 1);
	// Store v in the z velocity file

	// If compute_velocity = 1, we need to update dt
	if(compute_velocity == 1) {
		// CublasIdamax returns 1-indexed pointers into the max element of a
		// float-precision vector
		Updatedt<<<1,1>>>(cublasIsamax(N*(M-2), u, 1), u, cublasIsamax(N*(M-2), v, 1), v, dt);
	}

	// Place Dxxf into y
	cublasScopy(N*(M-2), f, 1, y, 1);
	Dxx(f, y);

	// y = y + Dzzf
	// place Dzzf into Tbuff
	cublasScopy(N*(M-2), f, 1, Tbuff, 1);
	Dzz(f, Tbuff);

	// Add the elements of y and Tbuff, storing in y.
	cublasSaxpy(N*(M-2), 1.0, Tbuff, 1, y, 1);

	// u = u.*DxT, where .* denotes elementwise multiplication
	// Perform the elentwise multiplication, storing in u
	ElemMultT<<<grid, block>>>(u,DxT);

	// y = y + u
	// Add y and u, storing the result in y
	cublasSaxpy(N*(M-2), 1.0, u, 1, y, 1);

	// u = DzT
	cublasScopy(N*(M-2), f, 1, u, 1);
	Dz(f, 0, u);

	// u = v.*u, where .* denotes elementwise multiplication.
	ElemMultT<<<grid, block>>>(u,v);

	// y = y + u
	cublasSaxpy(N*(M-2), 1.0, u, 1, y, 1);

	// copy into output
	cublasScopy(N*(M-2), y, 1, output, 1);
	return;
}


//-----------------------------------------------------------------------------
//=============================================================================
//								 ENTRY POINT
//=============================================================================
//-----------------------------------------------------------------------------

int main(void) {

//=============================================================================
//								Initialization
//=============================================================================
	printf("M = %d. ",M);
	printf("N = %d. ",N);
	printf("DX = %E. ",DX);
	printf("Ra = %E.\n",RA);
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	printf("Device 0: %s\n", devProp.name);
	hipGetDeviceProperties(&devProp, 1);
	printf("Device 1: %s\n", devProp.name);
	hipGetDeviceProperties(&devProp, 2);
	printf("Device 2: %s\n", devProp.name);
	hipSetDevice(2);  // Use the Fermi on Mark2

	hipblasStatus_t custat;  //intended to hold cublasInit messages
	printf("\nInitializing CUBLAS\n");
	cublasInit();  // Initialize cublas
	custat = hipblasStatus_t();
	if(custat == HIPBLAS_STATUS_SUCCESS) {
		printf("CUBLAS successfuly initialized\n");
	}
	else if(custat == HIPBLAS_STATUS_ALLOC_FAILED) {
		printf("CUBLAS could not be initialized\n");
	}

	printf("Configuring memory.\n");
	// initialize dt to the chosen start parameter
	float dt = DT_START;
	float* d_dt; // Device-side shadow of dt, as a one-element array.
	cublasAlloc(1, sizeof(float), (void**)&d_dt);
	float* h_dt = (float*)malloc(sizeof(float)); // for ease later
	h_dt[0] = dt;
	hipblasSetVector(1, sizeof(float), h_dt, 1, d_dt, 1);

	// Define T
	float* d_T;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_T);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

//	CUALLOC((M-2)*N, d_T);

	float* h_X = (float*)malloc(N*sizeof(float));
	float* h_Z = (float*)malloc((M-2)*sizeof(float));
	float* h_T = (float*)malloc(N*(M-2)*sizeof(float));

	// Set the values of X and Z
	for(int i = 0; i < N; i++) {
		h_X[i] = (i*XF + 0.0)/(N - 1.0);
	}

	// Some wonky indexing here to accomodate the fact that T has its top and
	// bottom rows chopped off.
	for(int i = 1; i < (M - 1); i++) {
		h_Z[i - 1] = (i + 0.0)/(M - 1.0);
	}
	// Initialize T, perturbing it slightly.
	for(int i = 0; i < M-2; i++) {
		for(int j = 0; j < N; j++) {
			h_T[i*N + j] = 1 - h_Z[i] + 0.01*sin(PI*h_Z[i])*cos((PI/XF)*h_X[j]);
			// For debugging purposes
			//h_T[i*N + j] = i + 1;
		}
	}

	// If INPUTT is set, overwrite h_T with the contents of input3d.bin
	// Read T from a file, if desired.
	if(INPUTT == 1) {
		uint8_t h_tbuff = 0;
		FILE *inputfile;
		inputfile = fopen("input2d.bin","r");
		for(int i = M-3; i >= 0; i--) {
			for(int j = 0; j < N; j++) {
				fread(&h_tbuff, 1, 1, inputfile);
				h_T[i*N + j] = h_tbuff;
				h_T[i*N + j] /= 255.0;
				if(h_T[i*N + j] < 0 || h_T[i*N + j] > 1) printf("Error in inputfile: value outside of range!\n");
			}
		}
		fclose(inputfile);
	}

        // If INPUTT32 is set, overwrite h_T with the contents of input3d.bin
        // Read T from a file, if desired.
        if(INPUTT32 == 1) {
                float h_tbuff = 0;
                FILE *inputfile;
                inputfile = fopen("input2d32.bin","r");
                for(int i = M-3; i >= 0; i--) {
                        for(int j = 0; j < N; j++) {
                                fread(&h_tbuff, 4, 1, inputfile);
                                h_T[i*N + j] = h_tbuff;
                        }
                }
                fclose(inputfile);
        }



	// Define the arrays necessary to calculate the Nusselt number.
	// These will be transferred to the GPU.
	float* h_ztop = (float*)malloc(4*N*sizeof(float));
	float* h_zbot = (float*)malloc(4*N*sizeof(float));


	for(int i = 0; i < 4*N; i++) {
		h_ztop[i] = 0.0;
		h_zbot[i] = 0.0;
	}

	for(int i = 0; i < N; i++ ) {
		// First row of h_ztop
		h_ztop[i] = 1-3*DX;
		// Second row of h_ztop
		h_ztop[i + N] = 1-2*DX;
		// Third row of h_ztop
		h_ztop[i + 2*N] = 1-DX;

		// Second row of h_zbot
		h_zbot[i + N] = h_Z[0];
		// Third row of h_zbot
		h_zbot[i + 2*N] = h_Z[1];
		// Fourth row of h_zbot
		h_zbot[i + 3*N] = h_Z[2];
	}

	// The bottom row of ztop is 1.0 and the top row of zbot is 0.0
	for(int i = 3*N; i < 4*N; i++) {
		h_ztop[i] = 1.0;
	}


	// Shadow T in GPU memory.  Although cublasSetMatrix assumes its input is
	// in column-major,
	hipblasSetVector(N*(M-2), sizeof(float), h_T, 1, d_T, 1);
	//free(h_X);
	//free(h_Z);

	float* d_omega;
	custat = cublasAlloc((M-2)*(N-2), sizeof(float), (void**)&d_omega);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_psi;
	custat = cublasAlloc((M-2)*N, sizeof(float), (void**)&d_psi);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_dsc;
	custat = cublasAlloc((M-2)*(M-2), sizeof(float), (void**)&d_dsc);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_dsr;
	custat = cublasAlloc((N-2)*(N-2), sizeof(float), (void**)&d_dsr);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_ei;
	custat = cublasAlloc((N-2)*(M-2), sizeof(float), (void**)&d_ei);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_tr;
	custat = cublasAlloc(N*M, sizeof(float), (void**)&d_tr);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_trnu;
	custat = cublasAlloc(N, sizeof(float), (void**)&d_trnu);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Coult not allocate memory.\n");

	float* d_ztop;
	custat = cublasAlloc(4*N, sizeof(float), (void**)&d_ztop);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Coult not allocate memory.\n");

	float* d_zbot;
	custat = cublasAlloc(4*N, sizeof(float), (void**)&d_zbot);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Coult not allocate memory.\n");

	float* d_nutop;
	custat = cublasAlloc(4*N, sizeof(float), (void**)&d_nutop);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Coult not allocate memory.\n");

	float* d_nubot;
	custat = cublasAlloc(4*N, sizeof(float), (void**)&d_nubot);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Coult not allocate memory.\n");

	// alternate declarations.
//	CUALLOC((M-2)*(N-2), d_omega);
//	CUALLOC((M-2)*N, d_psi);
//	CUALLOC((M-2)*(M-2), d_dsc);
//	CUALLOC((N-2)*(N-2), d_dsr);
//	CUALLOC((M-2), d_lambda);
//	CUALLOC((N-2), d_mu);
//	CUALLOC((N-2)*(M-2), d_ei);
//	CUALLOC(N*M, d_tr);
//	CUALLOC(N, d_trnu);
//	CUALLOC(4*N, d_ztop);
//	CUALLOC(4*N, d_zbot);
//	CUALLOC(4*N, d_nutop);
//	CUALLOC(4*N, d_nubot);

	// Initialize dsr, dsc, lambda, mu, ei, and trNu and copy them over
	float* h_dsc = (float*)malloc((M-2)*(M-2)*sizeof(float));
	float* h_dsr = (float*)malloc((N-2)*(N-2)*sizeof(float));
	float* h_lambda = (float*)malloc((M-2)*sizeof(float));
	float* h_mu = (float*)malloc((N-2)*sizeof(float));
	float* h_ei = (float*)malloc((M-2)*(N-2)*sizeof(float));
	float* h_tr = (float*)malloc(M*N*sizeof(float));
	float* h_trnu = (float*)malloc(N*sizeof(float));
	// Set the value of h_dsc
	for(int i = 0; i < M-2; i++) {
		for(int j = 0; j < M-2; j++) {
			h_dsc[(M-2)*i + j] = sqrt(2.0/(M-1.0))*sin((i+1.0)*(j+1.0)*PI/(M-1.0));
			// For debugging purposes.
			//if(i == j) h_dsc[(M-2)*i + j] = 1;
			//else h_dsc[(M-2)*i + j] = 0;
		}
	}

	// Set the value of h_dsr.
	for(int i = 0; i < N-2; i++) {
		for(int j = 0; j < N-2; j++) {
			h_dsr[(N-2)*i + j] = sqrt(2.0/(N-1.0))*sin((i+1.0)*(j+1.0)*PI/(N-1.0));
			// For debugging purposes.
			//if(i == j) h_dsr[(N-2)*i + j] = 1;
			//else h_dsr[(N-2)*i + j] = 0;
		}
	}

	// Initialize lambda and mu, which are used to compute ei.
	for(int i = 0; i < M-2; i++) {
		h_lambda[i] = 2.0*cos((i + 1.0)*PI/(M - 1.0)) - 2.0;
	}

	for(int i = 0; i < N-2; i++) {
		h_mu[i] = 2.0*cos((i + 1.0)*PI/(N - 1.0)) - 2.0;
	}
	// Compute ei from lambda and mu.
	// The elements of ei are inverted on the last step to replace later
	// divisions by multiplications.
	for(int i = 0; i < M-2; i++) {
		for(int j = 0; j < N-2; j++) {
			h_ei[(N-2)*i + j] = h_lambda[i] + h_mu[j];
			h_ei[(N-2)*i + j] = (h_ei[(N-2)*i + j])*(h_ei[(N-2)*i + j]);
			h_ei[(N-2)*i + j] = 1/(h_ei[(N-2)*i + j]);
		}
	}
	// Compute tr
	for(int i = 0; i < M; i++) {
		for(int j = 0; j < N; j++) {
			h_tr[N*i + j] = DX*DX/4.0;
			if(j>0 && j<(M-1)) h_tr[N*i + j] = DX2/2.0;
			if(i>0 && i<(N-1)) h_tr[N*i + j] = DX2/2.0;
			if(j>0 && j<(M-1) && i>0 && i<(N-1)) h_tr[N*i + j] = DX2;
		}
	}

	// Compute trnu
	for(int i = 1; i < N-1; i++) {
		h_trnu[i] = DX;
	}
	h_trnu[0] = DX/2.0;
	h_trnu[N-1] = DX/2.0;

	// Copy the completed data over.
	hipblasSetVector((M-2)*(M-2), sizeof(float), h_dsc, 1, d_dsc, 1);
	hipblasSetVector((N-2)*(N-2), sizeof(float), h_dsr, 1, d_dsr, 1);
	hipblasSetVector((M-2)*(N-2), sizeof(float), h_ei, 1, d_ei, 1);
	hipblasSetVector(M*N, sizeof(float), h_tr, 1, d_tr, 1);
	hipblasSetVector(N, sizeof(float), h_trnu, 1, d_trnu, 1);
	hipblasSetVector(4*N, sizeof(float), h_ztop, 1, d_ztop, 1);
	hipblasSetVector(4*N, sizeof(float), h_ztop, 1, d_nutop, 1);
	hipblasSetVector(4*N, sizeof(float), h_zbot, 1, d_zbot, 1);
	hipblasSetVector(4*N, sizeof(float), h_zbot, 1, d_nubot, 1);

	// Free the intermediate matrices (they are no longer needed host-side)
	//free(h_dsr);
	//free(h_dsc);
	//free(h_ei);
	//free(h_tr);


	float* d_u;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_u);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_v;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_v);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_xrk3;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_xrk3);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_yrk3;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_yrk3);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_zrk3;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_zrk3);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_y;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_y);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");



	float* d_Tbuff;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_Tbuff);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

	float* d_DxT;
	custat = cublasAlloc(N*(M-2), sizeof(float), (void**)&d_DxT);
	if(custat != HIPBLAS_STATUS_SUCCESS) printf("Could not allocate memory.\n");

//	CUALLOC(N*(M-2), d_xrk3);
//	CUALLOC(N*(M-2), d_yrk3);
//	CUALLOC(N*(M-2), d_zrk3);
//	CUALLOC(N*(M-2), d_y);
//	CUALLOC(N*(M-2), d_u);
//	CUALLOC(N*(M-2), d_v);
//	CUALLOC(N*(M-2), d_Tbuff);
//	CUALLOC(N*(M-2), d_DxT);
//	CUALLOC(N, d_onevec);

	// use d_T to define d_psi, d_u, d_v, and d_y = 0.
	cublasScopy(N*(M-2), d_T, 1, d_psi, 1);
	cublasSaxpy(N*(M-2), -1.0, d_psi, 1, d_psi, 1);
	cublasScopy(N*(M-2), d_psi, 1, d_u, 1);
	cublasScopy(N*(M-2), d_psi, 1, d_v, 1);
	cublasScopy(N*(M-2), d_psi, 1, d_y, 1);

	//initialize all intermediate matrices to T;
	cublasScopy(N*(M-2), d_T, 1, d_xrk3, 1);
	cublasScopy(N*(M-2), d_T, 1, d_yrk3, 1);
	cublasScopy(N*(M-2), d_T, 1, d_zrk3, 1);
	cublasScopy(N*(M-2), d_T, 1, d_Tbuff, 1);
	cublasScopy(N*(M-2), d_T, 1, d_DxT, 1);


// DEBUG
// temporary buffer for use in d_*rk3 stuff
float* d_temp;
cublasAlloc(N*(M-2), sizeof(float), (void**)&d_temp);
cublasScopy(N*(M-2), d_T, 1, d_temp, 1);

	printf("Begin timestep computation: \n");

//=============================================================================
//								Computation
//=============================================================================


	// Variable to store timing information
	// clock_t is defined in time.h
	clock_t timer = clock();

	// Begin timestep computation.  Every 50 timesteps, state will be saved to
	// disk
	double frames = 0.0;
	int tstep = 0;
	for(int c = STARTSTEP; c <= ENDSTEP; c++) {
		// Use SHORTG macro to call g succinctly.
		// x = g(T,0)
		// z = g(T + (dt/3)*x, 0)
		// z = g(T + (2*dt/3)*z, 1)
		// T = T + (dt/4)*(x + 3z)
		// Store the first part of RK3 in d_xrk3
		SHORTG(d_T, 1, d_xrk3, c);

		// add (dt/3)*d_xrk3 to T, store the result in T temporarily.
//		cublasSaxpy(N*(M-2), (dt/3.0), d_xrk3, 1, d_T, 1);
//DEBUG
cublasScopy(N*(M-2), d_T, 1, d_temp,  1);
cublasSaxpy(N*(M-2), (dt/3.0), d_xrk3, 1, d_temp, 1);
		// Compute d_yrk3 = g(T + (dt/3)*d_xrk3, 0) by using the updated T.
//		SHORTG(d_T, 0, d_yrk3);

//DEBUG
SHORTG(d_temp, 0,d_yrk3,0);

		// return d_T to its original state by subtracting (dt/3)*x
//		cublasSaxpy(N*(M-2), (-(dt/3.0)), d_xrk3, 1, d_T, 1);
		// Add (2*dt/3)*d_yrk3 to T, store the result in T temporarily.
//		cublasSaxpy(N*(M-2), (2.0*(dt/3)), d_yrk3, 1, d_T, 1);
//DEBUG
cublasScopy(N*(M-2), d_T, 1, d_temp,  1);
cublasSaxpy(N*(M-2), 2.0*(dt/3.0), d_yrk3, 1, d_temp, 1);
		// Compute d_zrk3 = g(T + (2*dt/3)*d_yrk3) by using the updated T.
//		SHORTG(d_T, 1, d_zrk3);

//DEBUG
SHORTG(d_temp, 0, d_zrk3,0);

		// return d_T to its original state by subtracting (2*dt/3)*d_yrk3
//		cublasSaxpy(N*(M-2), (-(2.0*(dt/3))), d_yrk3, 1, d_T, 1);
		// T+= (dt/4)*(x + 3z)
		// Add (dt/4)*d_xrk3 to d_T
//		cublasSaxpy(N*(M-2), (dt/4.0), d_xrk3, 1, d_T, 1);
		// Add 3*(dt/4)*d_zrk3 to d_T
//		cublasSaxpy(N*(M-2), (3.0*(dt/4.0)), d_zrk3, 1, d_T, 1);
//DEBUG
cublasSaxpy(N*(M-2), (dt/4.0), d_xrk3, 1, d_T, 1);
cublasSaxpy(N*(M-2), 3.0*(dt/4.0), d_zrk3, 1, d_T, 1);


		// update the value of dt (in host) from d_dt (in device)
		hipblasGetVector(1, sizeof(float), d_dt, 1, h_dt, 1);
		dt = h_dt[0];
		frames += dt;

		if(frames > FRAMESIZE) {
			tstep++;
			// CLOCKS_PER_SEC is defined as the number of clock cycles per
			// second and is variable along CPUs.
			printf("c: %d ", c);
			// Print s elapsed per timestep.
			printf("t: %Es ", (float)(clock()-timer)/(CLOCKS_PER_SEC*FRAMESIZE));
			timer = clock();
			printf("dt: %E. ", dt);

			// If PRINTNU is set, write a file that keeps track of
			// several Nu samples.
			printf("\n");
			if(PRINTNU == 1) {
				// Calculate the nusselt number throughout the array and save
				char sname[100] = "NU_" BSNAME ".bin";
				// Open a stream for writing Nu data
				FILE *nufile = fopen(sname,"a");
				float nunum = NusseltCompute(d_T, d_nutop, d_ztop, d_zbot, d_nubot, d_trnu);
				fprintf(nufile, "%.1f,", nunum);
				fclose(nufile);
			}


			// If PRINTT is set, write the temperature array to file.
			if(PRINTT == 1) {
				// Write d_T to the data file.  INVERTS ROW ORDERING
				char stringnum[25];
				char sname[100] = "T" BSNAME;
				sprintf(stringnum,"%d.bin",tstep);
				strcat(sname,stringnum);
				// Open a stream for writing T data
				FILE *tfile = fopen(sname,"w");
				hipblasGetVector((M-2)*N, sizeof(float), d_T, 1, h_T, 1);
				for(int i = M-3; i > -1; i--) {
					for(int j = 0; j < N; j++) {
						WriteT32(h_T[i*N + j], tfile);
					}
				}
				fclose(tfile);
			}
			frames = 0.0;
		}
	}

	printf("Done.");
	return(0);
}
